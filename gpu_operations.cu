#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <fstream>
#include <vector>
// Define the kernel function
__global__ void intensiveComputation(float *device_array, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        for (int i = 0; i < 400; i++) {
            for (int j = 0; j < 400; j++) {
                device_array[idx] = sinf(device_array[idx]) + cosf(device_array[idx]);
            }
        }
    }
}

int main() {
    const int N = 10;
    float *host_array, *device_array;
    const int num_iterations = 500;

    // Allocate memory on the host and device
    host_array = (float *)malloc(N * sizeof(float));
    hipMalloc((void **)&device_array, N * sizeof(float));

    // Initialize host array
    for (int i = 0; i < N; i++) {
        host_array[i] = i;
    }

    // CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0;
    std::vector<double> inferenceTimings;

    for (int iter = 0; iter < num_iterations; ++iter) {
        // Start timing
        hipEventRecord(start);

        // Transfer data from host to device
        hipMemcpy(device_array, host_array, N * sizeof(float), hipMemcpyHostToDevice);

        // Launch the intensive computation kernel
        int threadsPerBlock = 256;
        int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
        intensiveComputation<<<blocksPerGrid, threadsPerBlock>>>(device_array, N);

        // Transfer data from device back to host
        hipMemcpy(host_array, device_array, N * sizeof(float), hipMemcpyDeviceToHost);

        // Stop timing
        hipEventRecord(stop);
        hipEventSynchronize(stop);

        hipEventElapsedTime(&milliseconds, start, stop);
        inferenceTimings.push_back(milliseconds);
    }


    std::ofstream timingsFile("inference_timings.txt");
    if (timingsFile.is_open()) {
        for (const auto& timing : inferenceTimings) {
            timingsFile << timing << std::endl;
        }
        timingsFile.close();
        std::cout << "Inference timings stored in 'inference_timings.txt'." << std::endl;
    }
    else {
        std::cerr << "Unable to open file for writing." << std::endl;
    }
    // Cleanup
    free(host_array);
    hipFree(device_array);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}
